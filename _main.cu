#include "hip/hip_runtime.h"
/**
 * @desc: Keccak 4-round pre-image attack implementation
 *  ╦╔═┌─┐┌─┐┌─┐┌─┐┬┌─  ╔═╗┌─┐┬ ┬  ┬┌─┐┬─┐
 *  ╠╩╗├┤ │  │  ├─┤├┴┐  ╚═╗│ ││ └┐┌┘├┤ ├┬┘
 *  ╩ ╩└─┘└─┘└─┘┴ ┴┴ ┴  ╚═╝└─┘┴─┘└┘ └─┘┴└─
 * @language: C++/CUDA
 */

#include <hip/hip_runtime.h>

#include <iostream>
#include <stdlib.h>
#include <inttypes.h>
#include <unistd.h>
#include <sys/time.h>
#include <sys/types.h>

#include "keccak.h"
#include "solver.h"

int main(int argc, char **argv) {
    KeccakSolver keccakSolver;

    keccakSolverInit(&keccakSolver, argc, argv);
    keccakSolverLoop(&keccakSolver);
    keccakSolverExit(&keccakSolver);

    return 0;
}
